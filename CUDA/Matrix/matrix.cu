
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 100

__global__ void add( float (*a)[N], float (*b)[N], float (*c)[N] )
{
    int row = blockIdx.x;
    int col = blockIdx.y;
    if( row < N && col < N )
        c[row][col] = a[row][col] + b[row][col];
}


float a[N][N];
float b[N][N];
float c[N][N];

void initialize( )
{
    for( int row = 0; row < N; ++row ) {
        for( int col = 0; col < N; ++col ) {
            a[row][col] = 1.0;
            b[row][col] = 2.0;
        }
    }
}


void check_result( )
{
    for( int row = 0; row < N; ++row ) {
        for( int col = 0; col < N; ++col ) {
            if( fabs( c[row][col] - 3.0 ) > 1E-06 ) {
                printf( "FAILED!\n" );
                goto done;
            }
        }
    }
    printf( "PASSED!\n" );
done: ;
}


int main( void )
{
    float (*dev_a)[N], (*dev_b)[N], (*dev_c)[N];
    hipError_t result;

    initialize( );

    hipMalloc( (void **)&dev_a, N * N * sizeof(float) );
    hipMalloc( (void **)&dev_b, N * N * sizeof(float) );
    hipMalloc( (void **)&dev_c, N * N * sizeof(float) );

    hipMemcpy( dev_a, a, N * N * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * N * sizeof(float), hipMemcpyHostToDevice );
    dim3 grid(N, N);
    add<<<grid, 1>>>( dev_a, dev_b, dev_c );
    result = hipGetLastError( );
    if( result != hipSuccess ) {
        printf( "Failure invoking the add kernel! (%s)\n", hipGetErrorString( result ) );
    }
    hipMemcpy( c, dev_c, N * N * sizeof(float), hipMemcpyDeviceToHost );

    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    check_result( );

    return EXIT_SUCCESS;
}
