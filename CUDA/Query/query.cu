
/*
 * Based on a program from Chapter 3 of "CUDA by Example" by Jason Sanders and Edward Kandrot.
 * (C) 2011 by NVIDIA. ISBN=0-13-138768-5.
 */

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/driver_types.h"

static void display_boolean( int value )
{
    if( value )
        printf( "TRUE" );
    else
        printf( "FALSE" );
    printf( "\n" );
}


int main( void )
{
    hipDeviceProp_t prop;
    int count;
    hipError_t error_code;

    // hipInit( 0 );

    error_code = hipGetDeviceCount( &count );
    switch( error_code ) {
    case hipErrorInsufficientDriver:
    	 printf( "Unable to query devices; driver unavailable!\n" );
	 return EXIT_FAILURE;

    case hipErrorNoDevice:
    	 printf( "No devices found!\n" );
	 return EXIT_FAILURE;

    case hipSuccess:
    	 printf( "hipGetDeviceCount succeeded, %d devices found!\n", count );
	 break;
    }

    for( int i = 0; i < count; ++i ) {
        hipGetDeviceProperties( &prop, i );
        printf( "\n" );
        printf( "========\n" );
        printf( "Device %d\n", i );
        printf( "========\n" );
        printf( "--- General Information\n" );
        printf( "\tName: %s\n", prop.name );
        printf( "\tIntegrated GPU: " ); display_boolean( prop.integrated );
        printf( "\tCompute capability: %d.%d\n", prop.major, prop.minor );
        printf( "\tClock rate: %d\n", prop.clockRate );
        printf( "\tDevice copy overlap: " ); display_boolean( prop.deviceOverlap );
        printf( "\tKernel execution timeout: " ); display_boolean( prop.kernelExecTimeoutEnabled );

        printf( "\n--- Memory Information\n" );
        printf( "\tTotal global memory: %ld bytes\n", prop.totalGlobalMem );
        printf( "\tTotal constant memory: %ld bytes\n", prop.totalConstMem );
        printf( "\tMaximum memory pitch: %ld bytes\n", prop.memPitch );
        printf( "\tTexture alignment: %ld\n", prop.textureAlignment );
        printf( "\tCan map host memory: " ); display_boolean( prop.canMapHostMemory );

        printf( "\n--- Multiprocessor Information\n" );
        printf( "\tMultiprocessor count: %d\n", prop.multiProcessorCount );
        printf( "\tConcurrent kernels: " ); display_boolean( prop.concurrentKernels );
        printf( "\tShared memory per block: %ld bytes\n", prop.sharedMemPerBlock );
        printf( "\tRegisters per block: %d\n", prop.regsPerBlock );
        printf( "\tThreads in warp: %d\n", prop.warpSize );
        printf( "\tMaximum threads per block: %d\n", prop.maxThreadsPerBlock );
        printf( "\tMaximum block dimensions: (%d, %d, %d)\n",
                prop.maxThreadsDim[0],
                prop.maxThreadsDim[1],
                prop.maxThreadsDim[2] );
        printf( "\tMaximum grid dimensions: (%d, %d, %d)\n",
                prop.maxGridSize[0],
                prop.maxGridSize[1],
                prop.maxGridSize[2] );
        printf( "\n" );
    }
    return EXIT_SUCCESS;
}
