
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add( int a, int b, int *c )
{
    *c = a + b;
}

int main( void )
{
    int  c;
    int *dev_c;
    hipError_t result;

    result = hipMalloc( (void **)&dev_c, sizeof(int) );
    if( result != hipSuccess ) {
        printf( "Failure to allocate memory on the device!\n" );
    }

    add<<<1, 1>>>( 2, 7, dev_c );
    result = hipGetLastError( );
    if( result != hipSuccess ) {
        printf( "Failure invoking the add kernel! (%s)\n", hipGetErrorString( result ) );
    }

    result = hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost );
    if( result != hipSuccess ) {
        printf( "Failure to copy result from the device to the host!\n" );
    }

    hipFree( dev_c );
    printf( "c = %d\n", c );
    return EXIT_SUCCESS;
}
