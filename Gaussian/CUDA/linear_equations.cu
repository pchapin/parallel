#include "hip/hip_runtime.h"
/*!
 * \file   linear_equations.c
 * \brief  A gaussian elimination solver.
 * \author (C) Copyright 2014 by Peter C. Chapin <PChapin@vtc.vsc.edu>
 *
 * This is the serial version of the algorithm.
 */

#include <stdio.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "linear_equations.h"

//! Does the elimination step of reducing the system.
__global__ void elimination
  ( int size, floating_type *a, floating_type *b, floating_type *temp_array, int *status )
{
    int            i, j, k;
    floating_type  temp, m;

    for( i = 0; i < size - 1; ++i ) {

        // Find the row with the largest value of |a[j][i]|, j = i, ..., n - 1
        k = i;
        m = fabs( MATRIX_GET( a, size, i, i ) );
        for( j = i + 1; j < size; ++j ) {
            if( fabs( MATRIX_GET( a, size, j, i ) ) > m ) {
                k = j;
                m = fabs( MATRIX_GET( a, size, j, i ) );
            }
        }

        // Check for |a[k][i]| zero.
        if( fabs( MATRIX_GET( a, size, k, i ) ) <= 1.0E-6 ) {
            *status= -2;
            return;
        }

        // Exchange row i and row k, if necessary.
        if( k != i ) {
            for( j = 0; j < size; ++j ) temp_array[j] = MATRIX_GET( a, size, i, j );
            for( j = 0; j < size; ++j ) MATRIX_PUT( a, size, i, j, MATRIX_GET( a, size, k, j ) );
            for( j = 0; j < size; ++j ) MATRIX_PUT( a, size, k, j, temp_array[j] );

            // Exchange corresponding elements of b.
            temp = b[i];
            b[i] = b[k];
            b[k] = temp;
        }

        // Subtract multiples of row i from subsequent rows.
        for( j = i + 1; j < size; ++j ) {
            m = MATRIX_GET( a, size, j, i ) / MATRIX_GET( a, size, i, i );
            for( k = 0; k < size; ++k )
                MATRIX_PUT( a, size, j, k, MATRIX_GET( a, size, j, k ) - m * MATRIX_GET( a, size, i, k ) );
            b[j] -= m * b[i];
        }
    }
    *status = 0;
    return;
}


//! Does the back substitution step of solving the system.
static int back_substitution( int size, floating_type *a, floating_type *b )
{
    floating_type sum;
    int           i, j;

    for( i = size - 1; i >=0; --i ) {
        if( fabs( MATRIX_GET( a, size, i, i ) ) <= 1.0E-6 ) {
            return -2;
        }

        sum = b[i];
        for( j = i + 1; j < size; ++j ) {
            sum -= MATRIX_GET( a, size, i, j ) * b[j];
        }
        b[i] = sum / MATRIX_GET( a, size, i, i );
    }
    return 0;
}


extern "C" int gaussian_solve( int size, floating_type *a, floating_type *b )
{
    int            return_code;
    floating_type *dev_temp_array;
    int           *dev_return_code;
    floating_type *dev_a;
    floating_type *dev_b;

    hipMalloc( (void **)&dev_a, size * size * sizeof( floating_type ) );
    hipMalloc( (void **)&dev_b, size * sizeof( floating_type ) );
    hipMalloc( (void **)&dev_temp_array, size * sizeof( floating_type ) );
    hipMalloc( (void **)&dev_return_code, sizeof( int ) );

    hipMemcpy( dev_a, a, size * size * sizeof( floating_type ), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size * sizeof( floating_type ), hipMemcpyHostToDevice );

    elimination<<<1, 1>>>( size, dev_a, dev_b, dev_temp_array, dev_return_code );
    hipError_t result = hipGetLastError( );
    if( result != hipSuccess ) {
        printf( "Failure invoking the elimination kernel! (%s)\n", hipGetErrorString( result ) );
    }

    hipMemcpy( &return_code, dev_return_code, sizeof( int ), hipMemcpyDeviceToHost );
    if( return_code == 0 ) {
        hipMemcpy( a, dev_a, size * size * sizeof( floating_type ), hipMemcpyDeviceToHost );
        hipMemcpy( b, dev_b, size * sizeof( floating_type ), hipMemcpyDeviceToHost );
        return_code = back_substitution( size, a, b );
    }
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_temp_array );
    hipFree( dev_return_code );
    return return_code;
}
